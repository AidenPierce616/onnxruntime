#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "contrib_ops/cuda/sparse/block_mask.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

__global__ void ExpandMask(int* expanded_mask, const int* mask, int max_blocks,
                           int row_splits, int col_splits, bool causal) {
  const int output_cols = max_blocks * col_splits;
  if (threadIdx.x >= output_cols) {
    return;
  }

  int expanded_col = threadIdx.x;
  int expanded_row = blockIdx.x;

  int layout_id = blockIdx.y;

  // Let mask and expanded_mask point to the start of current layout.
  mask += layout_id * max_blocks * max_blocks;
  const int output_rows = max_blocks * row_splits;
  expanded_mask += layout_id * output_rows * output_cols;

  // Get mask value from the original mask.
  const int row = expanded_row / row_splits;
  const int col = expanded_col / col_splits;
  int value = mask[row * max_blocks + col];

  // Apply causal constraint.
  if (causal && expanded_col * row_splits > expanded_row * col_splits) {
    value = 0;
  }

  expanded_mask[expanded_row * output_cols + expanded_col] = value;
}

void ExpandBlockMask(hipStream_t stream, int* expanded_mask, const int* mask,
                     int num_layout, int max_blocks, int row_splits, int col_splits,
                     bool causal, int max_threads_per_block) {
  // Each block handle one row. For example, max_blocks=64, col_splits=2, then each block handle 128 elements.
  int output_cols = max_blocks * col_splits;
  int threads_per_block = (output_cols + 31) / 32 * 32;

  // Each thread handle one row. The kernel assumes that all rows can be handled in one block.
  if (threads_per_block > max_threads_per_block) {
    ORT_THROW("Threads per block is too large: max_blocks=", max_blocks, ", col_splits=", col_splits,
              ", max_threads_per_block=", max_threads_per_block);
  }

  const int output_rows = max_blocks * row_splits;
  dim3 gridSize(output_rows, num_layout, 1);
  ExpandMask<<<gridSize, threads_per_block, 0, stream>>>(
      expanded_mask, mask, max_blocks, row_splits, col_splits, causal);
}

__global__ void MaskToCSR(const int* mask, int* csr_row_indices, int* csr_col_indices, int num_rows, int num_cols) {
  int row = threadIdx.x;
  if (row >= num_rows) {
    return;
  }

  // Update input and output data pointers to the start of current head
  int head = blockIdx.x;
  mask += head * num_rows * num_cols;
  csr_row_indices += head * (num_rows + 1);
  csr_col_indices += head * num_rows * num_cols;

  int count = 0;
  for (int col = 0; col < num_cols; col++) {
    if (mask[row * num_cols + col] == 1) {
      count++;
    }
  }

  extern __shared__ int non_zero_counts[];
  non_zero_counts[threadIdx.x] = count;
  __syncthreads();

  // The first thread will calculate the accumulated partial sum of non-zero counts.
  if (row == 0) {
    for (int i = 1; i < num_rows; i++) {
      non_zero_counts[i] += non_zero_counts[i - 1];
    }
  }
  __syncthreads();

  // The starting index of current row in csr_col_indices
  int offset = (row == 0) ? 0 : non_zero_counts[row - 1];

  // Output row indices.
  csr_row_indices[row] = offset;
  if (row == 0) {
    // The first thread output the last element.
    csr_row_indices[num_rows] = non_zero_counts[num_rows - 1];
  }

  for (int col = 0; col < num_cols; col++) {
    if (mask[row * num_cols + col] == 1) {
      csr_col_indices[offset] = col;
      offset++;
    }
  }

  // Note that the remaining buffer in csr_col_indices are not filled with dummy value, but it's fine.
  // The last element of csr_row_indices is the total number of non-zero elements.
}

void ConvertMaskToCSR(hipStream_t stream,
                      const int* mask,       // input mask with shape (num_layout, num_rows, num_cols)
                      int num_layout,        // number of layouts
                      int num_rows,          // number of rows
                      int num_cols,          // number of columns
                      int* csr_row_indices,  // output CSR row indices
                      int* csr_col_indices,  // output CSR column indices
                      int max_threads_per_block) {
  int threads_per_block = (num_rows + 31) / 32 * 32;

  // Each thread handle one row. The kernel assumes that all rows of one head can be handled in one block.
  if (threads_per_block > max_threads_per_block) {
    ORT_THROW("num_rows is too large: num_rows=", num_rows, ", max_threads_per_block=", max_threads_per_block);
  }

  MaskToCSR<<<num_layout, threads_per_block, threads_per_block * sizeof(int), stream>>>(
      mask, csr_row_indices, csr_col_indices, num_rows, num_cols);
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
